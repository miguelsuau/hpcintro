
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void jacobi(double * uold, double * unew, double * f, int width, int height, double lambda2){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if(index < width*height){
		int M = width+2;
		int i = index + M + 1 + 2 * (index / width);

		unew[i] = 0.25 * (uold[i-1] + uold[i+1] + uold[i-M] + uold[i+M] + lambda2*f[i]);	
	}
}