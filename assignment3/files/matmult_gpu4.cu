
#include <hip/hip_runtime.h>
// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____           ____n____
//    |        |           |        |          |       |
//  m |    A   |   X    k  |    B   |  =    m  |   C   |
//    |        |           |        |          |       |
//    ---------            ---------           ---------

__global__ void m4(int m, int n, int k, double *A, double *B, double *C) {
  double sum1 = 0, sum2 = 0, sum3 = 0, sum4 = 0;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  i *= 4;
  if (i < m && j < n){
      for (int h = 0; h < k; h++) {
        sum1 += A[i*k + h] * B[h*n + j];
        sum2 += A[(i+1)*k + h] * B[h*n + j];
        sum3 += A[(i+2)*k + h] * B[h*n + j];
        C[(i+3)*n + j] += A[(i+3)*k + h] * B[h*n + j];
      }
  C[i*n + j] = sum1;
  C[(i+1)*n + j] = sum2;
  C[(i+2)*n + j] = sum3;
  C[(i+3)*n + j] = sum4;
  }
}


extern "C" {
    void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        // Initialize the output matrix with zeroes.
        hipMemset(d_C, 0, m*n * sizeof(double));
        dim3 BlockDim(16,16);
        dim3 NumBlocks((m/4-1)/16+1,((n-1)/16+1));
        m4<<<NumBlocks,BlockDim>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
